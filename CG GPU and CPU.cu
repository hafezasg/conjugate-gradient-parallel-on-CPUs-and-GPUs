#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "time.h"

#define IDX2C(i,j,ld) (((j)*(ld))+( i ))
void MultMatVec(float* S,float *A1D,float* P);

int im=151, jm=151,N; 
hipError_t        cudaStat;
hipblasStatus_t    stat;
hipblasHandle_t    handle;

int main(int argc, char* argv[]) {



  float **A,*B,*X;
  
  int  i=0,j=0,m=0,n=0;
  N=(im+1)*(jm+1);

  // Allocate
  A=(float**)calloc(N,sizeof(float *));
  for(i=0;i<N;i++){
    A[i]=(float*)calloc(N,sizeof(float));
  }
  X=(float *)calloc(N,sizeof(float));
  B=(float *)calloc(N,sizeof(float));
 
//INITIALIZE CUDA EVENTS
hipEvent_t start,stop;
float elapsedTime;

//CREATING EVENTS
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start,0);

  // Form Matrix A
  float dx2=1./im,dy2=1./jm;
  for (m = 0; m < N; m++){
    j=(int)(m/(im+1)); // For bounday condition
    i=m-(j*(im+1));
    
    if(i==0 || j==0 || i==im || j==jm){
      n=m;
      A[m][n]=1.0;
    }else{
      n=m;
      A[m][n] =-2*((1/dx2)+(1/dy2));

      n=m+1;
      A[m][n]=1./dx2;
      
      n=m-1;
      A[m][n]=1./dx2;
    
      n=m+im+1;
      A[m][n]=1./dy2;
  
      n=m-(im+1);
      A[m][n]=1./dy2;
    }
  }//if not on boundary

  // Form RHS
  float XL=1.,XR=2.,XD=2.,XU=1.;

  for (m = 0; m < N; m++){
    j=(int)(m/(im+1)); 
    i=m-(j*(im+1));
    if(i==0){
           B[m]=XL;
    }
    if(i==im){
          B[m]=XR;
    }
    if(j==0){
          B[m]=XD;
    }
    if(j==jm){
         B[m]=XU;
    }
    if(i==0 && j==0){
          B[m]=(XL+XD)/2;
    }
    if(i==0 && j==jm){
          B[m]=(XL+XU)/2;
    }
    if(i==im && j==0){
         B[m]=(XR+XD)/2;
    }
    if(i==im && j==jm){
          B[m]=(XR+XU)/2;
    }
  }

  //Conjugate gradient
  int k=0,kmax=10;
  float tol=10e-3,res=1.,Beta,Alpha;
  float *r,*P,*S,*Z,*A1D;
  
   r=(float *)calloc(N,sizeof(float));
   P=(float *)calloc(N,sizeof(float));
   S=(float *)calloc(N,sizeof(float));
   Z=(float *)calloc(N,sizeof(float));
   A1D=(float *)calloc(N*N,sizeof(float));

for(j=0;j<N;j++){
    for(i=0;i<N;i++){
        A1D[IDX2C(i,j,N)]=A[i][j];
    }
}


   for (m = 0; m < N; m++){
       X[m]=0.;
       r[m]=B[m];
       P[m]=r[m];
   }
      
   do{
     k+=1;

     
MultMatVec(S,A1D,P);
// printf("K: %d, S[10]: %le \n",S[10]);
 
     float PDr=0.0;
     for (m = 0; m < N; m++){
       PDr+=P[m]*r[m];
     }

     float PDS=0.0;
     for (m = 0; m < N; m++){
       PDS+=P[m]*S[m];
     }
     Alpha=PDr/PDS;

     for (m = 0; m < N; m++){
       X[m]=X[m]+Alpha*P[m];
       r[m]=r[m]-Alpha*S[m];
     }

     MultMatVec(Z,A1D,r);
  
     float PDZ=0.0;
     for (m = 0; m < N; m++){
       PDZ+=P[m]*Z[m];
     }
     Beta=-PDZ/PDS;
    
     for (m = 0; m < N; m++){
       P[m]=r[m]+Beta*P[m];
     }

     float res=0.0;
     for (m = 0; m < N; m++){
       res+=r[m]*r[m];
     }
     res=sqrt(res);

     
     printf("K: %d, X[4]: %le , res:%le , Alpha: %le , Beta : %le \n",k,X[4],res,Alpha,Beta);
     
   } while (abs(res)>tol && k<kmax);

   
	 //FINISH RECORDING
hipEventRecord(stop,0);
hipEventSynchronize(stop);

//CALCULATE ELAPSED TIME
hipEventElapsedTime(&elapsedTime,start,stop);

//DISPLAY COMPUTATION TIME
//cout<<"\n\nElapsed Time = "<<elapsedTime<<" ms";
    printf("CPUtime: %le  \n",elapsedTime); 

   free(r);
   free(P);
   free(A);
   free(X);
   free(B);

 
   return 0;
} 

//----------------------------------------------------------------
void MultMatVec(float* S,float *A1D,float* P) {
 


float* d_a;
float* d_x;
float* d_y;

cudaStat=hipMalloc((void**)&d_a,N*N*sizeof(*A1D));
cudaStat=hipMalloc((void**)&d_x,N*sizeof(*P));
cudaStat=hipMalloc((void**)&d_y,N*sizeof(*S));

stat=hipblasCreate(&handle);
stat=hipblasSetMatrix(N,N,sizeof(*A1D),A1D,N,d_a,N);
stat=hipblasSetVector(N,sizeof(*P),P,1,d_x,1);
stat=hipblasSetVector(N,sizeof(*S),S,1,d_y,1);

float al=1.0f;
float bet=0.0f;

stat=hipblasSgemv(handle,HIPBLAS_OP_N,N,N,&al,d_a,N,d_x,1,&bet,d_y,1);

stat=hipblasGetVector(N,sizeof(*S),d_y,1,S,1);


}
//----------------------------------------------------------------
